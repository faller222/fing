#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "util.h"
#include "hip/hip_runtime.h"
#include ""

#define CHUNK 1024
//#define SIZE_X 4194304
#define SIZE_X 1048576
#define MASK_SIZE 21
#define PARTE 4

__constant__ int mask_constant[MASK_SIZE];


void cudaCheck()
{
  hipError_t hipError_t;

  hipError_t = hipGetLastError();
  if(hipError_t != hipSuccess)
  {
    printf("  hipGetLastError() returned %d: %s\n", hipError_t, hipGetErrorString(hipError_t));
  }else{
	//printf("  todo ok\n" );
  }
}

#if PARTE == 3



__global__ void Kernel_Convolucion_Constante(int * inputArray, int* outputArray)
{
	int ratio = MASK_SIZE / 2;
	int pValor = 0;	
	int pos = threadIdx.x + blockIdx.x * blockDim.x;
	for (int j=0;j<MASK_SIZE;j++){
		if (((pos + j - ratio) >= 0) && ((pos + j - ratio) < SIZE_X)){
			 pValor = pValor + inputArray[pos + j - ratio] * mask_constant[j];	
		}
		
	}	
	atomicAdd(&outputArray[pos],pValor);
}

#endif

__global__ void Kernel_Convolucion_Shared(int * inputArray, int* outputArray, int* mask)

{
	__shared__ int arrayincom[CHUNK + MASK_SIZE - 1];

	int pos = threadIdx.x + blockIdx.x * blockDim.x;
	int ratio = MASK_SIZE / 2;



	if(threadIdx.x==0){
		if(pos == 0){
			for(int j=0;j<ratio;j++){
				arrayincom[j]=0;
			}
		}else{
			for(int j=0;j<ratio;j++){
				arrayincom[j]=inputArray[pos-ratio+j];
			}
		}
	}else if(threadIdx.x==CHUNK-1){
		if (pos==SIZE_X-1){
			for(int j=1;j<=ratio;j++){
				arrayincom[threadIdx.x+j+ratio]=0;
			}
		}else{
			for(int j=1;j<=ratio;j++){
					arrayincom[threadIdx.x+ratio+j]=inputArray[pos+j];
			}
		}
	}
		
	arrayincom[threadIdx.x+ratio]=inputArray[pos];
	
	 __syncthreads();

	int pValor = 0;	
	for (int j=0;j<MASK_SIZE;j++){
			 pValor = pValor + arrayincom[threadIdx.x+j] * mask[j];	
	}

	atomicAdd(&outputArray[pos],pValor);
}


__global__ void Kernel_Convolucion_Simple(int * inputArray, int* outputArray, int* mask)
{
	int ratio = MASK_SIZE / 2;
	int pValor = 0;	
	int pos = threadIdx.x + blockIdx.x * blockDim.x;
	for (int j=0;j<MASK_SIZE;j++){
		if (((pos + j - ratio) >= 0) && ((pos + j - ratio) < SIZE_X)){
			 pValor = pValor + inputArray[pos + j - ratio] * mask[j];	
		}
		
	}
	atomicAdd(&outputArray[pos],pValor);

}

void Convolucion_C(int * inputArray, int* ouputArray, int * mask)
{
        int i, j;
        for( i = 0; i<SIZE_X;i++)
        {   
                for( j =0; j<MASK_SIZE;j++)
                        {      
                                int position = i-(int)(MASK_SIZE/2) + j;
                                if(position>=0 && position<SIZE_X)
                                        ouputArray[i] += inputArray[position] * mask[j];
                        }       
        }
} 

int main() {
	int* inputArray = (int*)malloc(sizeof(int) * SIZE_X);
	int* outputArray = (int*)malloc(sizeof(int) * SIZE_X);
	int* outputArray_GPU = (int*)malloc(sizeof(int) * SIZE_X);
	int* mask = (int*)malloc(sizeof(int) * MASK_SIZE);
	int i;
	int * inputArray_k;
	int * outputArray_k;


#if PARTE != 3
	int* mask_k;	
#endif

//hipMalloc del array de entrada
//hipMalloc del array de salida
	size_t size = SIZE_X * sizeof(int);
	hipMalloc(&(inputArray_k),size);
	hipMalloc(&(outputArray_k),size);
	

	
	
#if PARTE != 3
	//hipMalloc de la mascara
	size_t sizeMask = MASK_SIZE * sizeof(int);
	hipMalloc(&mask_k,sizeMask);
#endif

	for(i =0; i<SIZE_X;i++)
	{
		inputArray[i] = i;
		outputArray[i] = 0;
	}		

	for(i =0; i<MASK_SIZE; i++)
	{
		mask[i] = 1;
	}

  clockStart();
	Convolucion_C(inputArray, outputArray, mask);
	clockStop("CPU");
               
	clockStart();
  ///////////////////////////////////////
	//copiar datos de entrada a la GPU
	///////////////////////////////////////
   hipMemcpy(inputArray_k,inputArray,size,hipMemcpyHostToDevice);	
	
   hipMemset(outputArray_k, 0, sizeof(int)*SIZE_X );

#if PARTE == 3
	 //copia a memoria constante   
   hipMemcpyToSymbol(HIP_SYMBOL(mask_constant), mask, MASK_SIZE*sizeof(int));
#else
	hipMemcpy(mask_k, mask, sizeof(int)*MASK_SIZE, hipMemcpyHostToDevice);
#endif
	hipDeviceSynchronize();
	clockStop("Tranferencias a host");
	clockStart();
	int cantBloques = SIZE_X / CHUNK + (SIZE_X % CHUNK == 0 ? 0 : 1);

	int tamGrid = cantBloques;
	int tamBlock = CHUNK;
#if PARTE == 3
	printf("Convolucion_Constante\n");
	Kernel_Convolucion_Constante<<<tamGrid, tamBlock>>>(inputArray_k, outputArray_k);
#else

#if PARTE == 4
	printf("Convolucion_Shared\n");

	Kernel_Convolucion_Shared<<<tamGrid, tamBlock>>>(inputArray_k, outputArray_k, mask_k);
#else
	printf("Convolucion_Simple\n");
	Kernel_Convolucion_Simple<<<tamGrid, tamBlock>>>(inputArray_k, outputArray_k, mask_k);
	hipFree(mask_k);
#endif
#endif	

	

	hipDeviceSynchronize();
	
	clockStop("GPU");
	cudaCheck();
	clockStart();

	hipMemcpy(outputArray_GPU,outputArray_k,size,hipMemcpyDeviceToHost);	
	
	hipFree(inputArray_k);
	hipFree(outputArray_k);
	
  ///////////////////////////////////////
	//traer salida de la GPU
	///////////////////////////////////////
        

	clockStop("Tranferencias host a CPU");	
	
	 if(equal_arrays(outputArray_GPU,outputArray, SIZE_X))
		printf("Enhorabuena");
	 else
		printf("Rayos y centellas");
	 

	 free(outputArray_GPU);
	 free(outputArray);
	 free(inputArray);

	 char character;
	 scanf("%c", &character);
	return 0;
}
