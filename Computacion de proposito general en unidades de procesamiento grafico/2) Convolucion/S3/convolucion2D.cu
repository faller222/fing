#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include "util.h"

#define CHUNK 8
/*
#define SIZE_Y 768
#define SIZE_X 1024*/

#define SIZE_Y 8
#define SIZE_X 8
#define MASK_SIZE 3

void cudaCheck(){
	hipError_t hipError_t;
	hipError_t = hipGetLastError();
	if(hipError_t != hipSuccess)  {
		printf("  hipGetLastError() returned %d: %s\n", hipError_t, hipGetErrorString(hipError_t));
	}else{
		//printf("  todo ok\n" );
	}
}

int getIdx(int x,int y,int sizeY){
	return sizeY*x+y;
}

__device__ int getIdxD(int x,int y,int sizeY){
	return sizeY*x+y;
}

__global__ void Kernel_Convolucion_Shared2D(int * inputArray, int* outputArray, int* mask){
	
	__shared__ int compartida[CHUNK + MASK_SIZE - 1][CHUNK + MASK_SIZE - 1];
	
	
	int Col= blockDim.x*blockIdx.x+threadIdx.x;
	int Row= blockDim.y*blockIdx.y+threadIdx.y;
	printf("Col %d - Row %d \n",Col,Row);
	
	int indx = getIdxD(Col,Row,SIZE_Y);
	
	printf("Col %d - Row %d - IDX: %d\n",Col,Row,indx);
	int radio = MASK_SIZE / 2;
	
	compartida[threadIdx.x][threadIdx.y]=inputArray[indx];

	__syncthreads();

	int parcial = 0;
	int pX = Col-radio;
	int pY = Row-radio;
	for (int i=0;i<MASK_SIZE;i++){
		for (int j=0;j<MASK_SIZE;j++){
			if(pX>=0 && pX<SIZE_X)
				if(pY>=0 && pY<SIZE_Y)
					parcial += inputArray[getIdxD(pX,pY,SIZE_Y)] * mask[getIdxD(i,j,MASK_SIZE)];
			pY++;
		}
		pY = Row-radio;
		pX++;
	}
	atomicAdd(&outputArray[indx],indx);
}


//Chequeada en 8*8 con excel
void Convolucion2D_C(int * inputArray, int* ouputArray, int * mask){
	int x,y,i,j;
	for( x = 0; x<SIZE_X;x++){  
		for( y = 0; y<SIZE_Y;y++){	
			for( i =0; i<MASK_SIZE;i++){
				for( j =0; j<MASK_SIZE;j++){
					int pX = x-(int)(MASK_SIZE/2) + i;
					int pY = y-(int)(MASK_SIZE/2) + j;
					if(pX>=0 && pX<SIZE_X)
						if(pY>=0 && pY<SIZE_Y)
							ouputArray[getIdx(x,y,SIZE_Y)] += inputArray[getIdx(pX,pY,SIZE_Y)] * mask[getIdx(i,j,MASK_SIZE)];
				} 
			}  
		}    
	}
} 

int main() {
	int* inputArray = (int*)malloc(sizeof(int) * SIZE_X * SIZE_Y);
	int* outputArray = (int*)malloc(sizeof(int) * SIZE_X * SIZE_Y);
	int* outputArray_GPU = (int*)malloc(sizeof(int) * SIZE_X * SIZE_Y);
	int* mask = (int*)malloc(sizeof(int) * MASK_SIZE * MASK_SIZE);
	int i;
	int * inputArray_k;
	int* outputArray_k;	
	int* mask_k;	


//#########################################
//######## RESERVA DE MEMORIA #############
//#########################################
	size_t size = SIZE_X * SIZE_Y * sizeof(int);
	size_t sizeMask = MASK_SIZE * MASK_SIZE * sizeof(int);
	
	hipMalloc(&(inputArray_k),size);
	hipMalloc(&(outputArray_k),size);
	hipMalloc(&mask_k,sizeMask);

//########## INICIALIZACION ###############
	for(i =0; i<SIZE_X;i++)	{
		for(int j =0; j<SIZE_Y;j++)	{
			int aux = getIdx(i,j,SIZE_Y);
			inputArray[aux] = (aux)%100;
			outputArray[aux] = 0;
		}
	}	

	for(i =0; i<MASK_SIZE*MASK_SIZE; i++)	{
		mask[i] = 1;
	}
	
//########### CORRE EN CPU ################
	clockStart();
	Convolucion2D_C(inputArray, outputArray, mask);
	clockStop("CPU");
	print_matrix2D(outputArray,SIZE_X,SIZE_Y);

//########### COPIA A GPU #################             	   
	clockStart();
	hipMemcpy(inputArray_k,inputArray,size,hipMemcpyHostToDevice);		
	hipMemset(outputArray_k, 0, sizeof(int)*SIZE_X );
	hipMemcpy(mask_k, mask, sizeof(int)*MASK_SIZE, hipMemcpyHostToDevice);
	hipDeviceSynchronize();
	clockStop("Tranferencias a host");
	
//########### CORRE EN GPU ################  	
	clockStart();
	
	
	int cantBloquesX = SIZE_X / CHUNK + (SIZE_X % CHUNK == 0 ? 0 : 1);
	int cantBloquesY = SIZE_Y / CHUNK + (SIZE_Y % CHUNK == 0 ? 0 : 1);
	
	//Configurar la grilla
	dim3 tamGrid (cantBloquesX, cantBloquesY); //Grid dimensión
	dim3 tamBlock(CHUNK, CHUNK); //Block dimensión
	
	Kernel_Convolucion_Shared2D<<<tamGrid, tamBlock>>>(inputArray_k, outputArray_k, mask_k);

	hipDeviceSynchronize();
	clockStop("GPU");
	cudaCheck();
	
//########### RETORNO A CPU ################ 
	clockStart();
	hipMemcpy(outputArray_GPU,outputArray_k,size,hipMemcpyDeviceToHost);	
	hipFree(inputArray_k);
	hipFree(outputArray_k);
	clockStop("Tranferencias host a CPU");	
	print_matrix2D(outputArray_GPU,SIZE_X,SIZE_Y);
	print_matrix2D(inputArray,SIZE_X,SIZE_Y);
	
//########### CHECK ALL OK ################# 	
	if(equal_arrays(outputArray_GPU,outputArray, SIZE_X*SIZE_Y))
		printf("Enhorabuena\n");
    else
		printf("Rayos y centellas\n");
	
	free(inputArray);
	free(outputArray_GPU);
	free(outputArray);
	
	return 0;
}
