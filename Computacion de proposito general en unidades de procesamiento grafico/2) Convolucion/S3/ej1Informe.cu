#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include "util.h"

void cudaCheck(){
  hipError_t hipError_t;

  hipError_t = hipGetLastError();
  if(hipError_t != hipSuccess)
  {
    printf("  hipGetLastError() returned %d: %s\n", hipError_t, hipGetErrorString(hipError_t));
  }else{
	//printf("  todo ok\n" );
  }
}

__global__ void Kernel_Convolucion_Simple(int * inputArray, int* outputArray, int* mask, int arraySize, int maskSize)
{
	int ratio = maskSize / 2;
	int pValor = 0;	
	int pos = threadIdx.x + blockIdx.x * blockDim.x;
	for (int j=0;j<maskSize;j++){
		if (((pos + j - ratio) >= 0) && ((pos + j - ratio) < arraySize)){
			 pValor = pValor + inputArray[pos + j - ratio] * mask[j];	
		}
		
	}
	atomicAdd(&outputArray[pos],pValor);
}

void Convolucion_C(int * inputArray, int* ouputArray, int * mask, int arraySize,int maskSize)
{
	int i, j;
	for( i = 0; i<arraySize;i++)
	{   
		for( j =0; j<maskSize;j++)
			{      
				int position = i-(int)(maskSize/2) + j;
				if(position>=0 && position<arraySize)
					ouputArray[i] += inputArray[position] * mask[j];
			}       
	}
} 

void ejecutar(int arraySize,int maskSize,int blockSize){
	printf("\nTamaño: array %d, Mask %d, Block %d\n",arraySize,maskSize,blockSize);

	int* inputArray = (int*)malloc(sizeof(int) * arraySize);
	int* outputArray = (int*)malloc(sizeof(int) * arraySize);
	int* outputArray_GPU = (int*)malloc(sizeof(int) * arraySize);
	int* mask = (int*)malloc(sizeof(int) * maskSize);
	int i;
	int * inputArray_k;
	int* outputArray_k;	
	int* mask_k;


//#########################################
//######## RESERVA DE MEMORIA #############
//#########################################
	size_t size = arraySize * sizeof(int);
	size_t sizeMask = maskSize * sizeof(int);
	hipMalloc(&(inputArray_k),size);
	hipMalloc(&(outputArray_k),size);
	hipMalloc(&mask_k,sizeMask);
	
//########## INICIALIZACION ###############
	for(i =0; i<arraySize;i++){
		inputArray[i] = i;
		outputArray[i] = 0;
	}		

	for(i =0; i<maskSize; i++){
		mask[i] = 1;
	}

//########### COPIA A GPU #################             
	clockStart();
	hipMemcpy(inputArray_k,inputArray,size,hipMemcpyHostToDevice);	
	hipMemset(outputArray_k, 0, sizeof(int)*arraySize );
	hipMemcpy(mask_k, mask, sizeof(int)*maskSize, hipMemcpyHostToDevice);
	hipDeviceSynchronize();
	clockStop("\tTranferencias a host");
	
	
//########### CORRE EN GPU ################  	
	clockStart();
	int cantBloques = arraySize / blockSize + (arraySize % blockSize == 0 ? 0 : 1);
	int tamGrid = cantBloques;
	Kernel_Convolucion_Simple<<<tamGrid, blockSize>>>(inputArray_k, outputArray_k, mask_k, arraySize, maskSize);

	hipDeviceSynchronize();
	
	clockStop("\tGPU");
	cudaCheck();
	
//########### RETORNO A CPU ################ 	
	clockStart();
	hipMemcpy(outputArray_GPU,outputArray_k,size,hipMemcpyDeviceToHost);	
	hipFree(inputArray_k);
	hipFree(outputArray_k);
	clockStop("\tTranferencias host a CPU");	
	
	free(inputArray);
	free(outputArray_GPU);
	free(outputArray);
}

int main() {
	ejecutar(1048576,7,64);
	ejecutar(1048576,7,128);
	ejecutar(1048576,7,256);
	ejecutar(1048576,7,512);
	ejecutar(1048576,7,1024);		
	ejecutar(4194304,7,64);
	ejecutar(4194304,7,128);
	ejecutar(4194304,7,256);
	ejecutar(4194304,7,512);
	ejecutar(4194304,7,1024);	
	
	return 0;
}
