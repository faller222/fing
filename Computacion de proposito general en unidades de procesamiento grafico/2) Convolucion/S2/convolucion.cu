#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "util.h"
#include "hip/hip_runtime.h"
#include ""

#define CHUNK 1024
#define SIZE_X 4194304

#define MASK_SIZE 7
#define PARTE 2

void cudaCheck()
{
  hipError_t hipError_t;

  hipError_t = hipGetLastError();
  if(hipError_t != hipSuccess)
  {
    printf("  hipGetLastError() returned %d: %s\n", hipError_t, hipGetErrorString(hipError_t));
  }else{
	//printf("  todo ok\n" );
  }
}

#if PARTE == 3

__global__ void Kernel_Convolucion_Constante(int * inputArray, int* outputArray)
{

}

#endif

__global__ void Kernel_Convolucion_Shared(int * inputArray, int* outputArray, int* mask)
{
	
}


__global__ void Kernel_Convolucion_Simple(int * inputArray, int* outputArray, int* mask)
{
	int ratio = MASK_SIZE / 2;
	int pValor = 0;	
	int pos = threadIdx.x + blockIdx.x * blockDim.x;
	for (int j=0;j<MASK_SIZE;j++){
		if (((pos + j - ratio) >= 0) && ((pos + j - ratio) < SIZE_X)){
			 pValor = pValor + inputArray[pos + j - ratio] * mask[j];	
		}
		
	}
	//outputArray[pos]=pValor;
	atomicAdd(&outputArray[pos],pValor);

}

void Convolucion_C(int * inputArray, int* ouputArray, int * mask)
{
        int i, j;
        for( i = 0; i<SIZE_X;i++)
        {   
                for( j =0; j<MASK_SIZE;j++)
                        {      
                                int position = i-(int)(MASK_SIZE/2) + j;
                                if(position>=0 && position<SIZE_X)
                                        ouputArray[i] += inputArray[position] * mask[j];
                        }       
        }
} 

int main() {
	int* inputArray = (int*)malloc(sizeof(int) * SIZE_X);
	int* outputArray = (int*)malloc(sizeof(int) * SIZE_X);
	int* outputArray_GPU = (int*)malloc(sizeof(int) * SIZE_X);
	int* mask = (int*)malloc(sizeof(int) * MASK_SIZE);
	int i;
	int * inputArray_k;
	int * outputArray_k;	
#if PARTE != 3
	int* mask_k;	
#endif

//hipMalloc del array de entrada
//hipMalloc del array de salida
	size_t size = SIZE_X * sizeof(int);
	hipMalloc(&(inputArray_k),size);
	hipMalloc(&(outputArray_k),size);
	

	
	
#if PARTE != 3
	//hipMalloc de la mascara
	size_t sizeMask = MASK_SIZE * sizeof(int);
	hipMalloc(&mask_k,sizeMask);
#endif

	for(i =0; i<SIZE_X;i++)
	{
		inputArray[i] = i;
		outputArray[i] = 0;
	}		

	for(i =0; i<MASK_SIZE; i++)
	{
		mask[i] = 1;
	}

  clockStart();
	Convolucion_C(inputArray, outputArray, mask);
	clockStop("CPU");
               
	clockStart();
  ///////////////////////////////////////
	//copiar datos de entrada a la GPU
	///////////////////////////////////////
   hipMemcpy(inputArray_k,inputArray,size,hipMemcpyHostToDevice);	
	
   hipMemset(outputArray_k, 0, sizeof(int)*SIZE_X );

#if PARTE == 3
	 //copia a memoria constante
#else
	hipMemcpy(mask_k, mask, sizeof(int)*MASK_SIZE, hipMemcpyHostToDevice);
#endif
	hipDeviceSynchronize();
	clockStop("Tranferencias a host");
	clockStart();
	int cantBloques = SIZE_X / CHUNK + (SIZE_X % CHUNK == 0 ? 0 : 1);

	int tamGrid = cantBloques;
	int tamBlock = CHUNK;
#if PARTE == 3
	Kernel_Convolucion_Constante<<<tamGrid, tamBlock>>>(inputArray_k, outputArray_k);
#else

#if PARTE == 4
	Kernel_Convolucion_Shared<<<tamGrid, tamBlock>>>(inputArray_k, outputArray_k, mask_k);
#else
	Kernel_Convolucion_Simple<<<tamGrid, tamBlock>>>(inputArray_k, outputArray_k, mask_k);
#endif
#endif	

	

	hipDeviceSynchronize();
	
	clockStop("GPU");
	cudaCheck();
	clockStart();

	hipMemcpy(outputArray_GPU,outputArray_k,size,hipMemcpyDeviceToHost);	
	
	hipFree(inputArray_k);
	hipFree(outputArray_k);
	hipFree(mask_k);
  ///////////////////////////////////////
	//traer salida de la GPU
	///////////////////////////////////////
        

	clockStop("Tranferencias host a CPU");	
	
	 if(equal_arrays(outputArray_GPU,outputArray, SIZE_X))
		printf("Enhorabuena");
	 else
		printf("Rayos y centellas");
	 

	 free(outputArray_GPU);
	 free(outputArray);
	 free(inputArray);

	 char character;
	 scanf("%c", &character);
	return 0;
}
