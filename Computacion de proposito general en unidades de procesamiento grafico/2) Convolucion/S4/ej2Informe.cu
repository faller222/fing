#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include "util.h"

#define MASK_SIZE 21

__constant__ int mask_constant[MASK_SIZE];

void cudaCheck()
{
  hipError_t hipError_t;

  hipError_t = hipGetLastError();
  if(hipError_t != hipSuccess)
  {
    printf("  hipGetLastError() returned %d: %s\n", hipError_t, hipGetErrorString(hipError_t));
  }else{
	//printf("  todo ok\n" );
  }
}

__global__ void Kernel_Convolucion_Constante(int * inputArray, int* outputArray, int arraySize,int maskSize)
{
	
	int ratio = maskSize / 2;
	int pValor = 0;	
	int pos = threadIdx.x + blockIdx.x * blockDim.x;
	for (int j=0;j<maskSize;j++){
		if (((pos + j - ratio) >= 0) && ((pos + j - ratio) < arraySize)){
			 pValor = pValor + inputArray[pos + j - ratio] * mask_constant[j];	
		}
	}
	atomicAdd(&outputArray[pos],pValor);
}


void ejecute(int arraySize,int maskSize, int blockSize){

	printf("\nSizes: array %d, Mask %d, Block %d\n",arraySize,maskSize,blockSize);

	int* inputArray = (int*)malloc(sizeof(int) * arraySize);
	int* outputArray = (int*)malloc(sizeof(int) * arraySize);
	int* outputArray_GPU = (int*)malloc(sizeof(int) * arraySize);
	int* mask = (int*)malloc(sizeof(int) * MASK_SIZE);
	int i;
	int* inputArray_k;
	int* outputArray_k;	

//#########################################
//######## RESERVA DE MEMORIA #############
//#########################################
	size_t size = arraySize * sizeof(int);
	hipMalloc(&(inputArray_k),size);
	hipMalloc(&(outputArray_k),size);

//########## INICIALIZACION ###############
	for(i =0; i<arraySize;i++)	{
		inputArray[i] = i;
		outputArray[i] = 0;
	}		

	for(i =0; i<MASK_SIZE; i++)	{
		mask[i] = 5;
	}

//########### COPIA A GPU #################  
	clockStart();
	hipMemcpy(inputArray_k,inputArray,size,hipMemcpyHostToDevice);	
	hipMemset(outputArray_k, 0, sizeof(int)*arraySize );	
	hipMemcpyToSymbol(HIP_SYMBOL(mask_constant), mask, MASK_SIZE*sizeof(int));
	hipDeviceSynchronize();
	clockStop("\tTranferencias a host");
	
//########### CORRE EN GPU ################ 

	clockStart();
	int cantBloques = arraySize / blockSize + (arraySize % blockSize == 0 ? 0 : 1);
	int tamGrid = cantBloques;
	int tamBlock = blockSize;

	Kernel_Convolucion_Constante<<<tamGrid, tamBlock>>>(inputArray_k, outputArray_k,arraySize,maskSize);
	hipDeviceSynchronize();
	clockStop("\tGPU");
	cudaCheck();
	
//########### RETORNO A CPU ################ 
	clockStart();
	hipMemcpy(outputArray_GPU,outputArray_k,size,hipMemcpyDeviceToHost);	
	hipFree(inputArray_k);
	hipFree(outputArray_k);
	clockStop("\tTranferencias host a CPU");	
	
	free(inputArray);
	free(outputArray_GPU);
	free(outputArray);

}




int main() {
	ejecute(4194304,5,256);
	ejecute(4194304,7,256);
	ejecute(4194304,21,256);
	ejecute(1048576,5,256);
	ejecute(1048576,7,256);
	ejecute(1048576,21,256);
	 	
	return 0;
}
