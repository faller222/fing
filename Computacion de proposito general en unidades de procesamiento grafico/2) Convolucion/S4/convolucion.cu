#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include "util.h"

#define CHUNK 1024
//#define SIZE_X 4194304
#define SIZE_X 1048576
#define MASK_SIZE 21
#define PARTE 7

#if (PARTE == 3)||(PARTE == 7)
//######### PARA MEMORIA CONSTANTE ######### 
__constant__ int mask_constant[MASK_SIZE];
#endif

void cudaCheck(){
	hipError_t hipError_t;
	hipError_t = hipGetLastError();
	if(hipError_t != hipSuccess)  {
		printf("  hipGetLastError() returned %d: %s\n", hipError_t, hipGetErrorString(hipError_t));
	}else{
		//printf("  todo ok\n" );
	}
}


#if PARTE == 2
__global__ void Kernel_Convolucion_Simple(int * inputArray, int* outputArray, int* mask){
	
	int radio = MASK_SIZE / 2;
	int parcial = 0;
	int indx = threadIdx.x + blockIdx.x * blockDim.x;

	int auxI= indx - radio;
	
	for (int i=0;i<MASK_SIZE;i++){
		if ((auxI >= 0) && (auxI < SIZE_X)){
			 parcial +=  inputArray[auxI] * mask[i];	
		}
		auxI++;
	}
	atomicAdd(&outputArray[indx],parcial);
}
#endif

#if PARTE == 3
__global__ void Kernel_Convolucion_Constante(int * inputArray, int* outputArray){
	
	int radio = MASK_SIZE / 2;
	int parcial = 0;	
	int indx = threadIdx.x + blockIdx.x * blockDim.x;
	for (int i=0;i<MASK_SIZE;i++){
		if (((indx + i - radio) >= 0) && ((indx + i - radio) < SIZE_X)){
			 parcial = parcial + inputArray[indx + i - radio] * mask_constant[i];	
		}
	}
	atomicAdd(&outputArray[indx],parcial);
}
#endif

#if PARTE == 4
__global__ void Kernel_Convolucion_Shared(int * inputArray, int* outputArray, int* mask){
	__shared__ int compartida[CHUNK + MASK_SIZE - 1];
	
	int indx = threadIdx.x + blockIdx.x * blockDim.x;
	int radio = MASK_SIZE / 2;
	
	if(threadIdx.x==0){//si estoy en el primer hilo del bloque
		if(indx == 0){//en el primer bloque
			for(int j=0;j<radio;j++){
				compartida[j]=0;//completo con 0
			}
		}else{//si no es el primero
			for(int j=0;j<radio;j++){
				compartida[j]=inputArray[indx-radio+j];//completo con vecinos
			}
		}
	}else if(threadIdx.x==CHUNK-1){//si estoy en el ultimo hilo del bloque
		if (indx==SIZE_X-1){//en el ultimo bloque
			for(int j=1;j<=radio;j++){
				compartida[threadIdx.x+j+radio]=0;//completo con 0
			}
		}else{//si no es el ultimo
			for(int j=1;j<=radio;j++){
					compartida[threadIdx.x+radio+j]=inputArray[indx+j];//completo con vecinos
			}
		}
	}
	
	compartida[threadIdx.x+radio]=inputArray[indx];

	__syncthreads();

	int parcial = 0;	
	for (int j=0;j<MASK_SIZE;j++){
		parcial = parcial + compartida[threadIdx.x+j] * mask[j];	
	}

	atomicAdd(&outputArray[indx],parcial);
}
#endif



#if PARTE == 5
__global__ void Kernel_Convolucion_SharedCache(int * inputArray, int* outputArray, int* mask){
	__shared__ int compartida[CHUNK];
	
	int indx = threadIdx.x + blockIdx.x * blockDim.x;
	int radio = MASK_SIZE / 2;

	compartida[threadIdx.x]=inputArray[indx];
	__syncthreads();

	int parcial = 0;
	int auxI= indx - radio;
	//int auxT= threadIdx.x - radio;
	for (int i=0;i<MASK_SIZE;i++){
		if (((auxI) >= 0) && ((auxI) < SIZE_X)){
			parcial += inputArray[auxI] * mask[i];
			/*if (((auxT) >= 0) && ((auxT) < CHUNK)){
				parcial += compartida[auxT] * mask[i];	
			}else{
				parcial += inputArray[auxI] * mask[i];
			}*/
		}
		//auxT++;
		auxI++;
	}
	atomicAdd(&outputArray[indx],parcial);
}
#endif

#if PARTE == 6
__global__ void Kernel_Convolucion_Shared2D(int * inputArray, int* outputArray, int* mask){
	__shared__ int compartida[CHUNK + MASK_SIZE - 1];
	
	int indx = threadIdx.x + blockIdx.x * blockDim.x;
	int radio = MASK_SIZE / 2;
	
	if(threadIdx.x==0){//si estoy en el primer hilo del bloque
		if(indx == 0){//en el primer bloque
			for(int j=0;j<radio;j++){
				compartida[j]=0;//completo con 0
			}
		}else{//si no es el primero
			for(int j=0;j<radio;j++){
				compartida[j]=inputArray[indx-radio+j];//completo con vecinos
			}
		}
	}else if(threadIdx.x==CHUNK-1){//si estoy en el ultimo hilo del bloque
		if (indx==SIZE_X-1){//en el ultimo bloque
			for(int j=1;j<=radio;j++){
				compartida[threadIdx.x+j+radio]=0;//completo con 0
			}
		}else{//si no es el ultimo
			for(int j=1;j<=radio;j++){
					compartida[threadIdx.x+radio+j]=inputArray[indx+j];//completo con vecinos
			}
		}
	}
	
	compartida[threadIdx.x+radio]=inputArray[indx];

	__syncthreads();

	int parcial = 0;	
	for (int j=0;j<MASK_SIZE;j++){
		parcial = parcial + compartida[threadIdx.x+j] * mask[j];	
	}

	atomicAdd(&outputArray[indx],parcial);
}
#endif


#if PARTE == 7
__global__ void Kernel_Convolucion_Shared_Compartida(int * inputArray, int* outputArray){
	__shared__ int compartida[CHUNK + MASK_SIZE - 1];
	
	int indx = threadIdx.x + blockIdx.x * blockDim.x;
	int radio = MASK_SIZE / 2;
	
	if(threadIdx.x==0){//si estoy en el primer hilo del bloque
		if(indx == 0){//en el primer bloque
			for(int j=0;j<radio;j++){
				compartida[j]=0;//completo con 0
			}
		}else{//si no es el primero
			for(int j=0;j<radio;j++){
				compartida[j]=inputArray[indx-radio+j];//completo con vecinos
			}
		}
	}else if(threadIdx.x==CHUNK-1){//si estoy en el ultimo hilo del bloque
		if (indx==SIZE_X-1){//en el ultimo bloque
			for(int j=1;j<=radio;j++){
				compartida[threadIdx.x+j+radio]=0;//completo con 0
			}
		}else{//si no es el ultimo
			for(int j=1;j<=radio;j++){
					compartida[threadIdx.x+radio+j]=inputArray[indx+j];//completo con vecinos
			}
		}
	}
	
	compartida[threadIdx.x+radio]=inputArray[indx];

	__syncthreads();

	int parcial = 0;	
	for (int j=0;j<MASK_SIZE;j++){
		parcial = parcial + compartida[threadIdx.x+j] *  mask_constant[j];	
	}

	atomicAdd(&outputArray[indx],parcial);
}
#endif

void Convolucion_C(int * inputArray, int* ouputArray, int * mask){
	int i, j;
	for( i = 0; i<SIZE_X;i++){   
		for( j =0; j<MASK_SIZE;j++){      
			int position = i-(int)(MASK_SIZE/2) + j;
			if(position>=0 && position<SIZE_X)
				ouputArray[i] += inputArray[position] * mask[j];
		}       
	}
} 

int main() {
	int* inputArray = (int*)malloc(sizeof(int) * SIZE_X);
	int* outputArray = (int*)malloc(sizeof(int) * SIZE_X);
	int* outputArray_GPU = (int*)malloc(sizeof(int) * SIZE_X);
	int* mask = (int*)malloc(sizeof(int) * MASK_SIZE);
	int i;
	int * inputArray_k;
	int* outputArray_k;	
#if PARTE != 3
	int* mask_k;	
#endif

//#########################################
//######## RESERVA DE MEMORIA #############
//#########################################
	size_t size = SIZE_X * sizeof(int);
	hipMalloc(&(inputArray_k),size);
	hipMalloc(&(outputArray_k),size);
#if PARTE != 3
	//hipMalloc de la mascara
	size_t sizeMask = MASK_SIZE * sizeof(int);
	hipMalloc(&mask_k,sizeMask);
#endif

//########## INICIALIZACION ###############
	for(i =0; i<SIZE_X;i++)	{
		inputArray[i] = i;
		outputArray[i] = 0;
	}		

	for(i =0; i<MASK_SIZE; i++)	{
		mask[i] = 5;
	}
	
//########### CORRE EN CPU ################
	clockStart();
	Convolucion_C(inputArray, outputArray, mask);
	clockStop("CPU");

//########### COPIA A GPU #################             	   
	clockStart();
   hipMemcpy(inputArray_k,inputArray,size,hipMemcpyHostToDevice);		
   hipMemset(outputArray_k, 0, sizeof(int)*SIZE_X );

#if (PARTE == 3)||(PARTE == 7)
	//copia a memoria constante   
	hipMemcpyToSymbol(HIP_SYMBOL(mask_constant), mask, MASK_SIZE*sizeof(int));
#else
	hipMemcpy(mask_k, mask, sizeof(int)*MASK_SIZE, hipMemcpyHostToDevice);
#endif
	hipDeviceSynchronize();
	clockStop("Tranferencias a host");
	
//########### CORRE EN GPU ################  	
	clockStart();
	int cantBloques = SIZE_X / CHUNK + (SIZE_X % CHUNK == 0 ? 0 : 1);
	int tamGrid = cantBloques;
	int tamBlock = CHUNK;
#if PARTE == 2
	Kernel_Convolucion_Simple<<<tamGrid, tamBlock>>>(inputArray_k, outputArray_k, mask_k);
#endif
#if PARTE == 3
	Kernel_Convolucion_Constante<<<tamGrid, tamBlock>>>(inputArray_k, outputArray_k);
#endif
#if PARTE == 4
	Kernel_Convolucion_Shared<<<tamGrid, tamBlock>>>(inputArray_k, outputArray_k, mask_k);
#endif	
#if PARTE == 5
	Kernel_Convolucion_SharedCache<<<tamGrid, tamBlock>>>(inputArray_k, outputArray_k, mask_k);
#endif
#if PARTE == 6
	Kernel_Convolucion_Shared2D<<<tamGrid, tamBlock>>>(inputArray_k, outputArray_k, mask_k);
#endif
#if PARTE == 7
	Kernel_Convolucion_Shared_Compartida<<<tamGrid, tamBlock>>>(inputArray_k, outputArray_k);
#endif


	hipDeviceSynchronize();
	clockStop("GPU");
	cudaCheck();
	
//########### RETORNO A CPU ################ 
	clockStart();
	hipMemcpy(outputArray_GPU,outputArray_k,size,hipMemcpyDeviceToHost);	
	hipFree(inputArray_k);
	hipFree(outputArray_k);
	clockStop("Tranferencias host a CPU");	
	
//########### CHECK ALL OK ################# 	
	if(equal_arrays(outputArray_GPU,outputArray, SIZE_X))
		printf("Enhorabuena\n");
    else
		printf("Rayos y centellas\n");
	
	free(inputArray);
	free(outputArray_GPU);
	free(outputArray);
	
	return 0;
}
