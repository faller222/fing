#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include <math.h>

#include <stdio.h>

#include "CImg.h"

using namespace cimg_library;

#define K 10
#define W 3
#define DIM_BLOQUE 32


__global__ void nlm_kernel(float* inputArray_GPU,float* outputArray_GPU, int width, int height)

{
	__shared__ int arrayincom[((K+W)*2+DIM_BLOQUE)*((K+W)*2+DIM_BLOQUE)];

	int x=blockIdx.x * blockDim.x +threadIdx.x;
	int y=blockIdx.y * blockDim.y +threadIdx.y;
	int pos = y*(blockDim.x*gridDim.x)+ x;

	int x_mas=x+(K+W);
	int x_menos=x-(K+W);
	int y_mas=y+(K+W);
	int y_menos=y-(K+W);

	if((x+blockIdx.x)<x_mas){
		arrayincom[(x+blockIdx.x)*(blockDim.x*gridDim.x)+y]=inputArray_GPU[(x+blockIdx.x)*(blockDim.x*gridDim.x)+y];
		if((y+blockIdx.y)<y_mas){
			//COPIO PUNTA BAJA
		}
		if((y-blockIdx.y)>y_menos){
			//COPIO PUNTA ALTA
		}
	}
	if((x-blockIdx.x)>x_menos){
		 // COPIO MEDIO
		if((y+blockIdx.y)<y_mas){
			//COPIO PUNTA BAJA
		}
		if((y-blockIdx.y)>y_menos){
			//COPIO PUNTA ALTA
		}
	}

	if((y+blockIdx.y)<y_mas){
			//COPIO MEDIO 
	}
	if((y-blockIdx.y)>y_menos){
			//COPIO PUNTA ALTA
	}
	arrayincom[threadIdx.x+ratio]=inputArray[pos];
	
	 __syncthreads();

	int pValor = 0;	
	for (int j=0;j<MASK_SIZE;j++){
			 pValor = pValor + arrayincom[threadIdx.x+j] * mask[j];	
	}

	atomicAdd(&outputArray[pos],pValor);
}


void nlm_Normal(float *input,float norma,int size)
{
	for(int i=0;i<size;i++){
		input[i] = input[i]/norma;
	}
}

void nlm_DesNormal(float *input,float norma,int size)
{
	for(int i=0;i<size;i++){
		input[i] = trunc(input[i]*norma);
	}
}

void nlm_ImprimirValores(float *input,int size)
{

	for(int i=0;i<size;i++){
		printf("%f  ",input[i]);
		if (i%1024==0)
			printf("\n");
	}
}

void nlm_CPU(float * inputImage, float* outputImage, int width, int height, float sigma)
{
	int largo_arreglo=width*height;
	float suma=0;
	float consta=0;
	float dist=0;
	float peso=0;

	for (int px = 0 ; px < width ; px++){
		for (int py = 0 ; py < height ; py++){
			for (int sx = px - K/2 ; sx < px + K/2 ; sx++){
				for (int sy = py - K/2 ; sy < py + K/2 ; sy++){
					if ((sx>0)&&(sx<width)&&(sy>0)&&(sy<height)){
						for (int wx = - W/2 ; wx < W/2 ; wx++){
							for (int wy = - W/2 ; wy < W/2 ; wy++){
								if (((px+wx>0)&&(px+wx<width)&&(py+wy>0)&&(py+wy<height))&&((sx + wx>0)&&(sx + wx<width)&&(sy + wy>0)&&(sy + wy<height)))
									dist += pow( (inputImage[(px + wx)*height + (py + wy)] - inputImage[(sx + wx)*height  +(sy + wy)]) ,2 );
							}
						}
						peso = exp(-dist/pow(sigma,2));
						suma += inputImage[sx*height+sy] * peso;
						consta += peso;
						dist=0;
						//peso=0;
					}
				}
			}

			outputImage[px*height + py] = suma / consta;
			suma=0;
			consta=0;
			
		}
	}
} 


int main()
{
    

	float sigma =0.9;
	
	CImg<float> image("img\\fing.pgm");
	float * img_matrix = image.data();


	CImg<float> imageOut("img\\fing.pgm");
	float * img_matrixOut = imageOut.data();
	
	int width=image.width();
	int height=image.height();
	int k=10;
	int w=3;

	size_t size = width*height*sizeof(float);
	size_t size2=((k+w)*2+height)*((k+w)*2+width)*sizeof(float);
	

	// Tenemos que normalizar ver nota al final de la letra de practico

	nlm_Normal(img_matrix,255.0,width*height);
	
	// GPU BEGIN

	float* inputArray_GPU;
	float* outputArray_GPU;
	hipMalloc(&(inputArray_GPU),size2);
	hipMalloc(&(outputArray_GPU),size);

	/////////////////////////////////////
	//copiar datos de entrada a la GPU
	///////////////////////////////////////
	hipMemset(outputArray_GPU, 0, size);
	hipMemset(inputArray_GPU, 0, size2);

	// Copiamos a memoria de GPU teniendo encuenta que es 2*(k+w) mas grande

	int offset=(k+w)*(width+(k+w)*2);
	for (int i=0; i<height; i++){
		offset+=(k+w);
		hipMemcpy(&inputArray_GPU[offset],&img_matrix[i*width],width,hipMemcpyHostToDevice);
		offset+=width+(k+w);
	}
    
	//Configurar la grilla
	dim3 tamGrid (width/32, height/32); //Grid dimensión
	dim3 tamBlock(32, 32); //Block dimensión
	//int tamBlock = 32*32;
	//int tamGrid = width*height/tamBlock;

	nlm_kernel<<<tamGrid, tamBlock>>>(inputArray_GPU, outputArray_GPU,k ,w ,width, height);
	



	// GPU END




	//

	//filtrar
		
	//nlm_CPU(img_matrix, img_matrixOut, image.width(), image.height(), 10, 5, sigma);
	


	//	...
	//
	
	//nlm_DesNormal(img_matrixOut,255.0,image.width()*image.height());
	//nlm_ImprimirValores(img_matrixOut,image.width()*image.height());

	CImgDisplay main_disp(image,"Fing");
	
	CImgDisplay main_disp1(imageOut,"FingNuevo");
		
	while (!main_disp.is_closed()) {
		main_disp.wait();
	}

	

	return 0;
}


