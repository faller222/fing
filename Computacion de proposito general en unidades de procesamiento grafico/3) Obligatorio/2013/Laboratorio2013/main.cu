#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include <math.h>

#include <stdio.h>

#include "CImg.h"

using namespace cimg_library;


void nlm_Normal(float *input,float norma,int size)
{
	for(int i=0;i<size;i++){
		input[i] = input[i]/norma;
	}
}

void nlm_DesNormal(float *input,float norma,int size)
{
	for(int i=0;i<size;i++){
		input[i] = trunc(input[i]*norma);
	}
}

void nlm_ImprimirValores(float *input,int size)
{

	for(int i=0;i<size;i++){
		printf("%f  ",input[i]);
		if (i%1024==0)
			printf("\n");
	}
}

void nlm_CPU(float * inputImage, float* outputImage, int width, int height, int K, int W, float sigma)
{
	int largo_arreglo=width*height;
	float suma=0;
	float consta=0;
	float dist=0;
	float peso=0;

	for (int px = 0 ; px < width ; px++){
		for (int py = 0 ; py < height ; py++){
			for (int sx = px - K/2 ; sx < px + K/2 ; sx++){
				for (int sy = py - K/2 ; sy < py + K/2 ; sy++){
					if ((sx>0)&&(sx<width)&&(sy>0)&&(sy<height)){
						for (int wx = - W/2 ; wx < W/2 ; wx++){
							for (int wy = - W/2 ; wy < W/2 ; wy++){
								if (((px+wx>0)&&(px+wx<width)&&(py+wy>0)&&(py+wy<height))&&((sx + wx>0)&&(sx + wx<width)&&(sy + wy>0)&&(sy + wy<height)))
									dist += pow( (inputImage[(px + wx)*height + (py + wy)] - inputImage[(sx + wx)*height  +(sy + wy)]) ,2 );
							}
						}
						peso = exp(-dist/pow(sigma,2));
						suma += inputImage[sx*height+sy] * peso;
						consta += peso;
						dist=0;
						//peso=0;
					}
				}
			}

			outputImage[px*height + py] = suma / consta;
			suma=0;
			consta=0;
			
		}
	}
} 


int main()
{
    
	float sigma =0.9;
	
	CImg<float> image("img\\fing.pgm");
	float * img_matrix = image.data();


	CImg<float> imageOut("img\\fing.pgm");
	float * img_matrixOut = imageOut.data();
	


	size_t size = image.width()*image.height()*sizeof(float);
	

	// Tenemos que normalizar ver nota al final de la letra de practico

	nlm_Normal(img_matrix,255.0,image.width()*image.height());
	
	

	//

	//filtrar
		
	nlm_CPU(img_matrix, img_matrixOut, image.width(), image.height(), 10, 5, sigma);
	


	//	...
	//
	
	nlm_DesNormal(img_matrixOut,255.0,image.width()*image.height());
	//nlm_ImprimirValores(img_matrixOut,image.width()*image.height());

	CImgDisplay main_disp(image,"Fing");
	
	CImgDisplay main_disp1(imageOut,"FingNuevo");
		
	while (!main_disp.is_closed()) {
		main_disp.wait();
	}

	

	return 0;
}


