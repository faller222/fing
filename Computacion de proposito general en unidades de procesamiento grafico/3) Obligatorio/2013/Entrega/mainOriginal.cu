#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include "CImg.h"

using namespace cimg_library;

#define K 10
#define W 3
#define DIM_BLOQUE 32
#define SIGMA 0.9

char path[20] = "img\\fing_xl.pgm";

__int64 ctr1 = 0, ctr2 = 0, freq = 0;

void clockStart(){
	QueryPerformanceCounter((LARGE_INTEGER *)&ctr1);
}

void clockStop(const char * str){
	
	QueryPerformanceCounter((LARGE_INTEGER *)&ctr2);
	QueryPerformanceFrequency((LARGE_INTEGER *)&freq);
	printf("%s --> %fs\n",str,(ctr2 - ctr1) * 1.0 / freq);
	
}



void cudaCheck()
{
  hipError_t hipError_t;

  hipError_t = hipGetLastError();
  if(hipError_t != hipSuccess)
  {
    printf("  hipGetLastError() returned %d: %s\n", hipError_t, hipGetErrorString(hipError_t));
  }else{
	//printf("  todo ok\n" );
  }
}



__global__ void nlm_kernel(float* inputArray_GPU,float* outputArray_GPU, int width, int height)

{
	__shared__ float arrayincom[((K/2+W/2)*2+DIM_BLOQUE)*((K/2+W/2)*2+DIM_BLOQUE)];

	int marco_medio=(int)K/2+(int)W/2;
	int marco=marco_medio*2;
	int dim_arraycom=marco_medio*2+DIM_BLOQUE;
	int dim_in_out_array=marco_medio*2+width;

	// COPIO LOS PIXELES QUE LE CORRESPONDEN AL HILO
	
	for(int i=threadIdx.x; i<blockDim.x+marco ; i+=blockDim.x){
		for(int j=threadIdx.y; j<blockDim.y+marco ; j+=blockDim.y){
			arrayincom[i+j*dim_arraycom]=inputArray_GPU[(i+blockIdx.x * blockDim.x)+(j+blockIdx.y * blockDim.y)*dim_in_out_array]; // Revisar bien los indices
		}
	}
	
	// SINCRONIZCO LOS HILOS PARA ASEGURARME QUE YA TENGO TODO CARGADO EN MEMORIA COMPARTIDA
	 __syncthreads();


	// EJECUTO NLM
	
	float suma=0;
	float consta=0;
	float dist=0;
	float peso=0;

	for (int sx = (threadIdx.x+marco_medio) - K/2 ; sx < (threadIdx.x+marco_medio) + K/2 ; sx++){
		for (int sy = (threadIdx.y+marco_medio) - K/2 ; sy < (threadIdx.y+marco_medio) + K/2 ; sy++){
				for (int wx = - W/2 ; wx < W/2 ; wx++){
					for (int wy = - W/2 ; wy < W/2 ; wy++){
							dist +=powf( (arrayincom[((threadIdx.x+marco_medio) + wx) + ((threadIdx.y+marco_medio) + wy)*dim_arraycom] - arrayincom[(sx + wx)  +(sy + wy)*dim_arraycom]) ,2 );
					}
				}
				peso = expf(-dist/powf(SIGMA,2));
				suma += arrayincom[sx+sy*dim_arraycom] * peso;
				consta += peso;
				dist=0;
			}
	}

	outputArray_GPU[(blockIdx.x * blockDim.x +threadIdx.x)+(blockIdx.y * blockDim.y +threadIdx.y)*width] =suma/consta;
	
}


void nlm_Normal(float *input,float norma,int size)
{
	for(int i=0;i<size;i++){
		input[i] = input[i]/norma;
	}
}

void nlm_DesNormal(float *input,float norma,int size)
{
	for(int i=0;i<size;i++){
		input[i] = input[i]*norma;
	}
}

void nlm_ImprimirValores(float *input,int size)
{

	for(int i=0;i<size;i++){
		printf("%f  ",input[i]);
		if (i%1024==0)
			printf("\n");
	}
}


void nlm_CPU(float * inputImage, float* outputImage, int width, int height)
{
	
	float suma=0;
	float consta=0;
	float dist=0;
	float peso=0;

	for (int px = 0 ; px < width ; px++){
		for (int py = 0 ; py < height ; py++){
			for (int sx = px - K/2 ; sx < px + K/2 ; sx++){
				for (int sy = py - K/2 ; sy < py + K/2 ; sy++){
					if ((sx>0)&&(sx<width)&&(sy>0)&&(sy<height)){
						for (int wx = - W/2 ; wx < W/2 ; wx++){
							for (int wy = - W/2 ; wy < W/2 ; wy++){
								if (((px+wx>0)&&(px+wx<width)&&(py+wy>0)&&(py+wy<height))&&((sx + wx>0)&&(sx + wx<width)&&(sy + wy>0)&&(sy + wy<height)))
									dist += pow( (inputImage[(px + wx) + (py + wy)*width] - inputImage[(sx + wx)  +(sy + wy)*width]) ,2 );
							}
						}
						peso = exp(-dist/pow(SIGMA,2));
						suma += inputImage[sx+sy*width] * peso;
						consta += peso;
						dist=0;
					}
				}
			}

			outputImage[py*width + px] = suma / consta;
			suma=0;
			consta=0;
			
		}
	}
} 


int main()
{


	/* CARGA IMAGEN */

	//CImg<float> image("img\\fing.pgm");

	CImg<float> image(path);
	float * img_matrix = image.data();
	
	CImg<float> imageOutGPU(path);
	float * img_matrixOutGPU = imageOutGPU.data();

	CImg<float> imageOutCPU(path);
	float * img_matrixOutCPU = imageOutCPU.data();

	/* FIN CARGA */

	int width=image.width();
	int height=image.height();
	int k=K/2;
	int w=W/2;

	size_t size = width*height*sizeof(float);
	size_t size2=((k+w)*2+height)*((k+w)*2+width)*sizeof(float);
	

	// Tenemos que normalizar ver nota al final de la letra de practico

	nlm_Normal(img_matrix,255.0,width*height);
	
	// GPU BEGIN

	float* inputArray_GPU;
	float* outputArray_GPU;
	hipMalloc(&(inputArray_GPU),size2);
	hipMalloc(&(outputArray_GPU),size);

	/////////////////////////////////////
	//copiar datos de entrada a la GPU
	///////////////////////////////////////
	hipMemset(outputArray_GPU, 0, size);
	
	hipMemset(inputArray_GPU, 0, size2);
	

	// Copiamos a memoria de GPU teniendo encuenta que es 2*(k+w) mas grande
	int offset=(k+w)*(width+(k+w)*2);
	
	for (int i=0; i<height; i++){
		offset+=(k+w);
		hipMemcpy(&inputArray_GPU[offset],&img_matrix[i*width],width*sizeof(float),hipMemcpyHostToDevice);
		offset+=width+(k+w);
	}
	

	//Configurar la grilla
	dim3 tamGrid (width/DIM_BLOQUE, height/DIM_BLOQUE); //Grid dimensión
	dim3 tamBlock(DIM_BLOQUE, DIM_BLOQUE); //Block dimensión

	clockStart();	
	
	nlm_kernel<<<tamGrid, tamBlock>>>(inputArray_GPU, outputArray_GPU,width,height);
	
	hipDeviceSynchronize();

	clockStop("GPU");
	cudaCheck();
	
	hipMemcpy(img_matrixOutGPU,outputArray_GPU,size,hipMemcpyDeviceToHost);
	
	
	nlm_DesNormal(img_matrixOutGPU,255.0,imageOutGPU.width()*imageOutGPU.height());


	hipFree(outputArray_GPU);
	// GPU END

	
	// CPU BEGIN


	clockStart();	

	nlm_CPU(img_matrix, img_matrixOutCPU, image.width(), image.height());
	
	clockStop("CPU");

	nlm_DesNormal(img_matrixOutCPU,255.0,imageOutCPU.width()*imageOutCPU.height());

	// CPU END
	
	
	

	CImgDisplay main_disp(image,"Fing");
	
	CImgDisplay main_disp1(imageOutGPU,"Fing - GPU");

	CImgDisplay main_disp2(imageOutCPU,"Fing - CPU");
		
	while (!main_disp.is_closed()) {
		main_disp.wait();
	}

	return 0;
}


