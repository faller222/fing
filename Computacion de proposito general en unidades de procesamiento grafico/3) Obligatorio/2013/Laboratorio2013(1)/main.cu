﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include "CImg.h"

using namespace cimg_library;

int main()
{
    
	float sigma =0.9f;

	CImg<float> image("img\\fing.pgm");
	
	float * img_matrix = image.data();

	size_t size = image.width()*image.height()*sizeof(float);
	
	//filtrar
	//	...
	//

	CImgDisplay main_disp(image,"Fing");
		
	while (!main_disp.is_closed()) {
		main_disp.wait();
	}

	return 0;
}
