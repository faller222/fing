#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include <math.h>

#include <stdio.h>

#include "CImg.h"

using namespace cimg_library;


void nlm_CPU(float * inputImage, float* outputImage, int width, int height, int K, int W, float sigma)
{
	int largo_arreglo=width*height;
	float suma=0;
	float consta=0;
	float dist=0;

	for (int px = 0 ; px < width ; px++){
		for (int py = 0 ; py < height ; py++){
			for (int sx = px - K/2 ; sx < px + K/2 ; sx++){
				for (int sy = py - K/2 ; sy < py + K/2 ; sy++){
					if ((sx>0)&&(sx<largo_arreglo)&&(sy>0)&&(sy<largo_arreglo)){
						for (int wx = - W/2 ; wx < W/2 ; wx++){
							for (int wy = - W/2 ; wy < W/2 ; wy++){
								if ((wx>0)&&(wx<largo_arreglo)&&(wy>0)&&(wy<largo_arreglo))
									dist += pow( (inputImage[(px + wx)*height + (py + wy)] - inputImage[(sx + wx)*height  +(sy + wy)]) ,2 );
							}
						}
						suma += inputImage[sx*height+sy] * exp(-dist/pow(sigma,2));
						consta += exp(-dist/pow(sigma,2));
						dist=0;
					}
				}
			}

			outputImage[px*height + py] = suma / consta;
			suma=0;
			consta=0;
		}
	}

    /*    int i, j;
        for( i = 0; i<SIZE_X;i++)
        {   
                for( j =0; j<MASK_SIZE;j++)
                        {      
                                int position = i-(int)(MASK_SIZE/2) + j;
                                if(position>=0 && position<SIZE_X)
                                        ouputArray[i] += inputArray[position] * mask[j];
                        }       
        }*/
} 


int main()
{
    
	float sigma =0.9f;

	CImg<float> image("img\\fing.pgm");

	//CImg<float> imageout("img\\fing.pgm");
	
	float * img_matrix = image.data();


	size_t size = image.width()*image.height()*sizeof(float);
	
	//filtrar

	float * img_matrixOut=(float*)malloc(size);


	nlm_CPU(img_matrix, img_matrixOut, image.width(), image.height(), 10, 3, sigma);

	//	...
	//
	//CImg<float> imageOut(img_matrixOut);
	
	CImgDisplay main_disp(image,"Fing");
	//CImgDisplay main_disp1(imageOut,"FingNuevo");
		
	while (!main_disp.is_closed()) {
		main_disp.wait();
	}

	return 0;
}


