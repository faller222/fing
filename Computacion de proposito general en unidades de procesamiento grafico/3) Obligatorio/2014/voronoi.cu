#include "hip/hip_runtime.h"
#ifndef KERNEL_CUDA
#define KERNEL_CUDA

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include "CImg.h"
#include "utilImage.h"

#define HILOS 32


using namespace cimg_library;
/**
Referencia:
	http://en.wikipedia.org/wiki/CUDA
	http://inf.ufrgs.br/gppd/wsppd/2013/papers/wsppd2013_submission_15.pdf.mod.pdf
*/

/**
	Variables en el kernel
		gridDim // cantidad de bloques
		blockDim //tamaño de bloque, cant de hilos
		threadIdx //inidice de hilo en el bloque, max blockDim-1
		blockIdx //indice de bloque , max gridDim-1 
*/

void cudaCheck(){
	hipError_t hipError_t = hipGetLastError();
	if(hipError_t != hipSuccess)  {
		printf("  hipGetLastError() returned %d: %s\n", hipError_t, hipGetErrorString(hipError_t));
		exit(0);
	}
}

__global__ void Kernel_Test(){
	printf("\tSay hi form BK(%d,%d) TH(%d,%d)\n",blockIdx.x,blockIdx.y,threadIdx.x,threadIdx.y);
}

__global__ void Kernel_Imagen_Promedio(float* inputArray_k, float* outputArray_k){

	__shared__ float frame[(HILOS + MARCO + MARCO)*(HILOS + MARCO + MARCO)];	
	
	int sizeX = blockDim.x * gridDim.x;
	//int sizeY = blockDim.y * gridDim.y;
	
	int id_hilo = blockDim.x * threadIdx.y + threadIdx.x;
	int cant_hilos = blockDim.y * blockDim.x; 
	
	int size_sh_x = HILOS + MARCO + MARCO;
	int size_shared = size_sh_x * size_sh_x;
	
	int offsetX = blockDim.x * blockIdx.x;
	int offsetY = blockDim.y * blockIdx.y;
	
	int sizeX_marco = sizeX + 2 * MARCO;
	
	int shX;
	int shY;
	
	for (int i = id_hilo ; i < size_shared ; i+=cant_hilos){		
		shX = (i % size_sh_x) + offsetX;
		shY = (i / size_sh_x) + offsetY;
		frame[i] = inputArray_k[ shX  + sizeX_marco * shY];		
	}
	
	__syncthreads();
	
	int acu = 0;
	//#pragma unroll
	for (int y = threadIdx.y ; y < threadIdx.y + SIZE_MASK ; y++){			
		//#pragma unroll
		for (int x = threadIdx.x ; x < threadIdx.x + SIZE_MASK ; x++){ 
			acu += (unsigned char) frame[x + size_sh_x*y];
		}
	}
	acu = acu /(SIZE_MASK * SIZE_MASK);
	outputArray_k[(threadIdx.x + offsetX) + sizeX * (threadIdx.y + offsetY)] =  acu;
}

__global__ void Kernel_Voronoi(float* inputArray_k, float* outputArray_k, bool* cambiosMatriz_k, int* indicesArrayO_k, int* indicesArrayU_k, int* fin){

	//indices
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	int y = blockDim.y * blockIdx.y + threadIdx.y;
	int sizeX = blockDim.x * gridDim.x;
	
	int tid = x + sizeX * y;
	
	int coord;
	
	int aux_dist;
	int nid;
	
	if(cambiosMatriz_k[tid]){
		if(indicesArrayO_k[tid]==-1){
			indicesArrayO_k[tid]=tid;
			indicesArrayU_k[tid]=tid;
		}
		
		//para cada vecino			
		for(int i=(x-1); i<=(x+1);i++){			
			for(int j=(y-1); j<=(y+1);j++){
				
				//dentro de los limites de la imagen
				if( i>=0 && i<sizeX &&  j>=0 && j<(blockDim.y * gridDim.y) ){
					nid = i + sizeX * j;
					if(indicesArrayO_k[nid]==-1){
						indicesArrayU_k[nid]=indicesArrayO_k[tid];
					}else{
						if(indicesArrayO_k[nid]!=indicesArrayO_k[tid]){
							
							//calculo de indices vecino
							//CoordX vecino
							coord = indicesArrayO_k[nid] % sizeX;												
							aux_dist= i * i + coord * coord - 2 * i * coord ;
							//CoordY vecino
							coord = indicesArrayO_k[nid] / sizeX;	
							aux_dist+= j * j + coord * coord - 2 * j * coord ;
							//Calculo de indices Actual
							//CoordX Actual
							coord = indicesArrayO_k[tid] % sizeX;
							aux_dist-= i * i + coord * coord - 2 * i * coord ;
							//CoordY Actual
							coord = indicesArrayO_k[tid] / sizeX;	
							aux_dist-= j * j + coord * coord - 2 * j * coord ;
												
							if(aux_dist > 0){
								indicesArrayU_k[nid]=indicesArrayO_k[tid];
							}
						}
					}
				}
			}
		}
		cambiosMatriz_k[tid]=false;
	}
	
	__syncthreads();
	
	if(indicesArrayU_k[tid]!=indicesArrayO_k[tid]){
		indicesArrayO_k[tid]=indicesArrayU_k[tid];
		cambiosMatriz_k[tid]=true;
		atomicAdd(&fin[0],1);
	}
	indicesArrayU_k[tid]=indicesArrayO_k[tid];
	
	outputArray_k[tid]=inputArray_k[indicesArrayU_k[tid]];	
}


CImg<float> voronoi_CPU(CImg<float> orig, CImg<float> &suave, int cantCentros){
	//Hacemos pading y generamos un recuadro con espejo
	CImg<float> p1  = prepararImagen(orig);

	//tamanios Originales
	int size_x = orig.width();
	int size_y = orig.height();
	//tamanios sin Padding
	int sizeX = p1.width() - SIZE_MASK + 1;
	int sizeY = p1.height() - SIZE_MASK + 1;
	// generamos Centros
	bool* centrosBool = generarCentros(cantCentros,sizeX, sizeY);
	//suavizado
	suave = roundImagen(p1);
	//voronoi
	CImg<float> p3 = voronoi(suave, cantCentros, centrosBool);
	free(centrosBool);
	suave=recortar(suave, size_x, size_y);
	return recortar(p3, size_x, size_y);
}

CImg<float> voronoi_GPU(CImg<float> orig, CImg<float> &suave, int cantCentros){	
	//Hacemos pading y generamos un recuadro con espejo
	CImg<float> p1  = prepararImagen(orig);
	float* data=p1.data();
	
	//Variables timming
	timeAux start;
	double time;
	//Variables en el Kernel
	float* inputArray_k;
	float* suaveArray_k;	
	float* voronArray_k;	
	bool* cambiosMatriz_k;
	int* indicesArrayO_k;
	int* indicesArrayU_k;
	int* finVoronoi;

	//tamanios originales
	int size_x_orig = orig.width();
	int size_y_orig = orig.height();
	//tamanios sin Padding
	int sizeX = p1.width() - SIZE_MASK + 1;
	int sizeY = p1.height() - SIZE_MASK + 1;
	//tamanios Kernel
	size_t sizeI = p1.width() * p1.height() * sizeof(float);
	size_t sizeO = sizeX * sizeY * sizeof(float);
	size_t sizeC = sizeX * sizeY * sizeof(bool);
	size_t sizeD = sizeX * sizeY * sizeof(int);
	
	//Resultados
	CImg<float> suavei(sizeX,sizeY,1,1);
	float* suave_data=suavei.data();
	CImg<float> voro(sizeX,sizeY,1,1);
	float* voro_data=voro.data();
	
	// generamos Centros
	bool* centros = generarCentros(cantCentros, sizeX, sizeY);
	
//######## RESERVA DE MEMORIA #############
	start = clockStart();
	hipMalloc(&(inputArray_k),sizeI);
	hipMalloc(&(suaveArray_k),sizeO);
	hipMalloc(&(voronArray_k),sizeO);
	hipMalloc(&(cambiosMatriz_k),sizeC);
	hipMalloc(&(indicesArrayO_k),sizeD);
	hipMalloc(&(indicesArrayU_k),sizeD);
	hipMalloc(&(finVoronoi),sizeof(int));
	time = clockStop(start);	
	printf("Reserva Memoria en el Device (%fs)\n",time);

//########### COPIA A GPU #################   
	start = clockStart(); 	  
	hipMemcpy(inputArray_k,data,sizeI,hipMemcpyHostToDevice);		
	hipMemset(suaveArray_k, 0, sizeO );
	hipMemset(voronArray_k, 0, sizeO );
	hipMemcpy(cambiosMatriz_k,centros,sizeC,hipMemcpyHostToDevice);
	hipMemset(indicesArrayO_k,-1,sizeD);
	hipMemset(indicesArrayU_k,-1,sizeD);
	hipDeviceSynchronize();
	time = clockStop(start);
	printf("Tranferencia Host -> Device (%fs)\n",time);
	
//####### Configurar la grilla ############  
	start = clockStart(); 	
	int cantBloquesX = sizeX / HILOS;
	int cantBloquesY = sizeY / HILOS;
	dim3 tamGrid (cantBloquesX, cantBloquesY);
	dim3 tamBlock(HILOS, HILOS );

//############# CORRE EN GPU ##############  
	printf("Ejecutando Suavizado...  ");	 
	Kernel_Imagen_Promedio<<<tamGrid, tamBlock>>>(inputArray_k, suaveArray_k);
	hipDeviceSynchronize();
	time = clockStop(start);
	printf("Listo (%fs)\n",time);
	
	start = clockStart(); 
	printf("Ejecutando Voronoi...  ");
	int iteraciones=0;
	int* fin =(int*)malloc(sizeof(int));
	fin[0]=-4;		
	
	while(fin[0]){
		hipMemset(finVoronoi,0,sizeof(int));
		Kernel_Voronoi<<<tamGrid, tamBlock>>>(suaveArray_k, voronArray_k,  cambiosMatriz_k,indicesArrayO_k,indicesArrayU_k,finVoronoi);
		hipDeviceSynchronize();
		hipMemcpy(fin,finVoronoi,sizeof(int),hipMemcpyDeviceToHost);	
		cudaCheck();
		iteraciones++;
	}
	time = clockStop(start);
	printf("Listo (%fs) %d Iteraciones\n",time,iteraciones);
	
	
//########### RETORNO A CPU ################ 
	start = clockStart(); 	
	//suavizado
	hipMemcpy(suave_data,suaveArray_k,sizeO,hipMemcpyDeviceToHost);
	//Voronoi
	hipMemcpy(voro_data,voronArray_k,sizeO,hipMemcpyDeviceToHost);
	time = clockStop(start);
	printf("Tranferencia Device -> Host(%fs)\n",time);
	
//########### LIBERA MEMORIA ############### 	
	
	hipFree(inputArray_k);
	hipFree(suaveArray_k);
	hipFree(voronArray_k);
	hipFree(cambiosMatriz_k);
	hipFree(indicesArrayO_k);
	hipFree(indicesArrayU_k);
	hipFree(finVoronoi);
	free(centros);

	suave=recortar(suavei, size_x_orig, size_y_orig);
	return recortar(voro, size_x_orig, size_y_orig);
}

#endif