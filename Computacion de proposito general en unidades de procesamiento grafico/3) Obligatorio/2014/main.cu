#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include "CImg.h"
#include "utilImage.h"
#include "voronoi.cu"

using namespace cimg_library;

int main(int argc, char** argv){
	timeAux start = clockStart(); 
	bool CPU= false;
	int cual =  1 ;
	int cantCentros =  30000;
	char* name = "img\/fing.pgm";
	if(argc>1){
		cual =  atoi(argv[1]);		
	}else{
		printf ("1 - CPU\n2 - GPU\n_: ");
		scanf ("%d",&cual);
		CPU = (cual==1);

		printf ("Que imagen? \n\t1-L\n\t2-XL\n\t3-XXL\n\t_: ");
		scanf ("%d",&cual);  
		printf ("Ingrese la cantidad de centros: ");
		scanf ("%d",&cantCentros);
		start = clockStart(); 
	}

	if(cual==2){
		name = "img\/fing_xl.pgm";
	}else{
		if(cual==3){
			name = "img\/fing_xxl.pgm";
		}else{
			cantCentros =  30000;
		}
	}


	if(argc>2){
		cantCentros =  atoi(argv[2]);
	}
	if(argc>3){
		CPU= true;
	}
	
	CImg<float> orig(name);
	CImg<float> suave;
	CImg<float> voro;
	if(CPU){
		voro = voronoi_CPU(orig,suave,cantCentros);
		//guardarImagenC(voro,"CPU - P2 - Voronoi.png");
	}else{
		voro = voronoi_GPU(orig,suave,cantCentros);
		//guardarImagenC(voro,"GPU - P2 - Voronoi.png");
	}
	double time = clockStop(start);	
	printf("Todo el proceso (%fs)\n",time);

#if defined (_WIN32)
	CImgDisplay main_dispOri(orig,"Original");
	CImgDisplay main_dispSuave(suave,"Suavizada");
	CImgDisplay main_dispVoro(voro,"Voronoi");
	while (!(main_dispVoro.is_closed()||main_dispSuave.is_closed()||main_dispOri.is_closed()));
#else
	guardarImagenC(suave,"Suavizada.png");
	guardarImagenC(voro,"Voronoi.png");
#endif	
	
	
	return 0;
}
