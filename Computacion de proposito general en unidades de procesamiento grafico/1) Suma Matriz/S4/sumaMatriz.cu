#include "hip/hip_runtime.h"
#include "util.h"

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""


using namespace std;

#define TILE_WIDTH 16


void clockStart();	
void clockStop(const char * str);

unsigned long long int sum_matrix(const unsigned long long int *M, int width);
void print_matrix(const unsigned long long int *M, int width);
void clean_matrix(unsigned long long int *M, int width);
void init_matrix(unsigned long long int *M, int width);

//Kernel

// Suma por columnas de una matriz con un solo bloque
__global__ void MatrixSumKernel_1(int M, unsigned long long int* Md, unsigned long long int* Nd){

// Pvalue es usado para el valor intermedio
  unsigned long long int Pvalue = 0;
  
  int offset = threadIdx.y * M;
  
  for (int k = 0; k < M; k++) {
     Pvalue = Pvalue + Md[offset+k];
  }

  Nd[threadIdx.y] = Pvalue;

}

// Fila de bloques (cada bloque suma varias columnas) - cada trhead suma una columna entera
__global__ void MatrixSumKernel_2(int M, unsigned long long int* Md, unsigned long long int* Nd){

	// calcular id global
	// sumar columna
}

// Un bloque suma varias columnas - cada thread suma parte de la columna, utilizar atomicAdd para sumar
__global__ void MatrixSumKernel_3(int M,int N,unsigned long long int* Md, unsigned long long int* Nd){

	// Pvalue es usado para el valor intermedio
	unsigned long long int Pvalue = 0;
	
	// calcular id global
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	
	int pos = blockIdx.y * (N/gridDim.y);
		
	int pasos = N/gridDim.y;
	
	// Pvalue = sumar(...)
	for (int k = 0 ;k<pasos;k++){
		Pvalue = Pvalue + Md[ col*M + pos + k];
	}

	atomicAdd(&(Nd[0]), Pvalue);
}

//extern "C" 
unsigned long long int sumaColMatriz(int M, int N, unsigned long long int * Mh, int algoritmo){


	switch(algoritmo){
		case 1:
		{
			size_t size = M * N * sizeof(unsigned long long int);
			size_t size2 = N*sizeof(unsigned long long int);

			unsigned long long int* Md, *Nd;

			unsigned long long int *Nh = (unsigned long long int *)malloc(N*sizeof(unsigned long long int));

			// Allocate en device 
			hipMalloc(&Md, size);
			hipMalloc(&Nd, size2);

			// Inicializo matrices en el device
			hipMemcpy(Md, Mh, size, hipMemcpyHostToDevice);
			hipMemset(Nd,0, size2);

			//Configurar la grilla
			dim3 tamGrid (1, 1); //Grid dimensión
			dim3 tamBlock(1, N); //Block dimensión

			MatrixSumKernel_1<<<tamGrid, tamBlock>>>(M, Md, Nd);

			// Traer resultado;
			hipMemcpy(Nh, Nd, size2, hipMemcpyDeviceToHost);

			// Sumar el vector de resultados parciales;
			unsigned long long int total = 0.0;
			for (int i = 0; i<N ; i++) total += Nh[i];

			// Free matrices en device
			hipFree(Md); hipFree(Nd); 

			return total;

		}
		case 2:
		{
			// Fila de bloques (cada bloque suma varias columnas) - cada trhead suma una columna entera
			printf("\n\nNo implementado aún!! :)\n\n\n");

			//...
			break;
		}		
		case 3:
		{
			// Un bloque por columna - cada thread suma parte de la columna, utilizar atomicAdd para sumar
			size_t size = M * N * sizeof(unsigned long long int);
			size_t size2 = N*sizeof(unsigned long long int);

			unsigned long long int* Md, *Nd;

			unsigned long long int *Nh = (unsigned long long int *)malloc(N*sizeof(unsigned long long int));

			// Allocate en device 
			hipMalloc(&Md, size);
			hipMalloc(&Nd, size2);

			// Inicializo matrices en el device
			hipMemcpy(Md, Mh, size, hipMemcpyHostToDevice);
			hipMemset(Nd,0, size2);

			
			//Configurar la grilla
			dim3 tamGrid (N/1024, 32); //Grid dimensión
			dim3 tamBlock(1024, 1); //Block dimensión
			clockStart();
			MatrixSumKernel_3<<<tamGrid, tamBlock>>>(M,N, Md, Nd);
			clockStop("GPU Sin Transferencia");
			// Traer resultado;
			hipMemcpy(Nh, Nd, size2, hipMemcpyDeviceToHost);

			// Sumar el vector de resultados parciales;
			unsigned long long int total = 0.0;
			for (int i = 0; i<N ; i++) total += Nh[i];

			// Free matrices en device
			hipFree(Md); hipFree(Nd); 

			return total;
			

			//...
			break;
		}
	}

}


int main(int argc, char** argv){

	if (argc < 3){
		printf("Uso:\nMatSum n algo(1:3)");
		exit(0);
	}
	
	int n= atoi(argv[1]);
	int algo = atoi(argv[2]);	
	
	unsigned long long int *A = (unsigned long long int *)malloc(n*n*sizeof(unsigned long long int));

	init_matrix(A,n);

	clockStart();
	unsigned long long int result_ref = sum_matrix(A,n);
	clockStop("CPU");
	printf("algo - %i \n",algo);
	clockStart();
	unsigned long long int result_gpu = sumaColMatriz(n,n,A,algo);
	clockStop("GPU");
	if (result_gpu == result_ref)
		printf("\n\nResultado OK!! :)\n\n\n");
	else
		printf("\n\Segui participando\n\n\n");

	free(A);	
	


	return 0;
}


