#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include "util.h"

using namespace std;

#define chunk 256


void clockStart();	
void clockStop(const char * str);


float sum_matrix(const float *M,  int width);
void print_matrix(const float *M,  int width);
void clean_matrix(float *M,  int width);
void init_matrix(float *M,  int width);

//Kernel

// Suma por columnas de una matriz con un solo bloque
__global__ void MatrixSumKernel_0(int M, float* A_dev, float* SumPar_dev){

	// Pvalue es usado para el valor intermedio
	double Pvalue = 0;
  
	int offset = threadIdx.y * M;
  
	for (int k = 0; k < M; k++) {
		Pvalue = Pvalue + A_dev[offset+k];
	}
	
	SumPar_dev[threadIdx.y] = Pvalue;

}

__global__ void MatrixSumKernel_1(int M, float* A_dev, float* SumPar_dev){
	// Pvalue es usado para el valor intermedio
	float Pvalue = 0;
  
	int columna = blockIdx.x;
	int fCol = columna * gridDim.x;
  
	for (int k = 0; k < M; k++) {
		Pvalue = Pvalue + A_dev[fCol+k];
	}
	
	SumPar_dev[blockIdx.x] = Pvalue;
}

__global__ void MatrixSumKernel_2(int M,float* A_dev, float* SumPar_dev){

	float parcial = 0;
	int nCol = blockIdx.y;
	int fCol = nCol * gridDim.y;
	
	int pasos = M/ blockDim.x;
	int step = fCol + threadIdx.x*pasos;

	if(threadIdx.x<M){
		for (int k = 0; k < pasos; ++k) {
			parcial = parcial + A_dev[step + k];
		}
		atomicAdd(&(SumPar_dev[nCol]), parcial);
	}
}

__global__ void MatrixSumKernel_3(int M,float* A_dev, float* SumPar_dev){

	int pasos;	
	if(blockDim.x>M){
		pasos = 1;
	}else{
		pasos = M / blockDim.x;
	}	
		
	float parcial = 0;
	int nCol = blockIdx.y;
	int fCol = nCol * gridDim.y;
	int step = fCol + threadIdx.x;
	
	if(threadIdx.x<M){
		for (int k = 0; k < pasos; ++k) {
			parcial = parcial + A_dev[step + k*blockDim.x];
		}
		atomicAdd(&(SumPar_dev[nCol]), parcial);
	}
}

__global__ void MatrixSumKernel_4(int M,float* A_dev, float* SumPar_dev){
	
	// Cant Filas por bloque
	int rowXblock = M / gridDim.y;
	
	// Cant Columnas por bloque
	int colXblock = M / gridDim.x;
	
	//Cant Columnas que le conrresponden a un thread
	int colXthread = colXblock / blockDim.x ;	
	
	
	float parcial;
	int nCol;
	int paso;
	
	for (int j = 1; j <= colXthread; ++j) {
		parcial = 0;
		nCol = blockIdx.x * colXblock + threadIdx.x * j;
		paso = nCol * M + rowXblock * blockIdx.y;
		
		for (int k = 0; k < rowXblock; ++k) {
			parcial = parcial + A_dev[paso + k];
		}
		atomicAdd(&(SumPar_dev[nCol]), parcial);
	}
}

__global__ void MatrixSumKernel_5(int M,float* A_dev, float* SumPar_dev){
	
	// Cant Filas por bloque
	int rowXblock = M / gridDim.y;
	
	// Cant Columnas por bloque
	int colXblock = M / gridDim.x;
	
	//Cant Filas que le conrresponden a un thread
	int rowXthread = rowXblock / blockDim.x ;	
	
	float parcial;
	int nCol;
	int paso;
	
	for (int j = 0; j < colXblock; ++j) {
		parcial = 0;
		nCol = blockIdx.x * colXblock + j;
		paso = nCol * M + rowXblock * blockIdx.y;
		
		//For que suma la columna
		for (int k = 0; k < rowXthread; ++k) {
			int step = paso + threadIdx.x + k * blockDim.x;
			parcial = parcial + A_dev[step];
		}
		atomicAdd(&(SumPar_dev[nCol]), parcial);
	}
}

__global__ void MatrixSumKernel_6(int M,float* A_dev, float* SumPar_dev){
	
	extern __shared__ float Nds[];
	
	int pasos;	
	if(blockDim.x>M){
		pasos = 1;
	}else{
		pasos = M / blockDim.x;
	}	
		
	float parcial = 0;
	int nCol = blockIdx.y;
	int fCol = nCol * gridDim.y;
	int step = fCol + threadIdx.x;
	
	if(threadIdx.x<M){
		for (int k = 0; k < pasos; ++k) {
			parcial = parcial + A_dev[step + k*blockDim.x];
		}
	}else{
		parcial = 0;
	}
	Nds[threadIdx.x] = parcial;
	
	__syncthreads(); 
	
	if (threadIdx.x == 0){
		for (int i = 1; i < blockDim.x; ++i) {
			Nds[0] = Nds[0]+Nds[i];
		}
	SumPar_dev[nCol] = Nds[0];
	}
}

//extern "C" 
float sumaColMatriz(int M, int N, float * A_hst, int algoritmo){

	size_t size = M * N * sizeof(float);
	size_t size2 = N*sizeof(float);

	float* A_dev, *SumPar_dev;

	float *SumPar_hst = (float *)malloc(N*sizeof(float));

	// Allocate en device 
	hipMalloc(&A_dev, size);
	hipMalloc(&SumPar_dev, size2);

	// Inicializo matrices en el device
	//clockStart();
	hipMemcpy(A_dev, A_hst, size, hipMemcpyHostToDevice);
	hipMemset(SumPar_dev,0, size2);
	//clockStop("transf CPU -> GPU");

	clockStart();

	switch(algoritmo){
		case 0:{
			//Configurar la grilla
			dim3 tamGrid (1, 1); //Grid dimensi�n
			dim3 tamBlock(1, N); //Block dimensi�n

			MatrixSumKernel_0<<<tamGrid, tamBlock>>>(M, A_dev, SumPar_dev);

			hipDeviceSynchronize();
			clockStop("kernel 0");
			
			break;
		}case 1:{
			//Configurar la grilla
			dim3 tamGrid (N, 1); //Grid dimensi�n
			dim3 tamBlock(1, 1); //Block dimensi�n

			MatrixSumKernel_1<<<tamGrid, tamBlock>>>(M, A_dev, SumPar_dev);
			hipDeviceSynchronize();
			clockStop("kernel 1");
			
			break;
		}case 2:{
			// configuraci�n de la ejecuci�n
			int chunk2 = 32;
			//Configurar la grilla
			dim3 tamGrid (1, N); //Grid dimensi�n
			dim3 tamBlock(chunk2,1, 1); //Block dimensi�n

			MatrixSumKernel_2<<<tamGrid, tamBlock>>>(M, A_dev, SumPar_dev);
			hipDeviceSynchronize();
			clockStop("kernel 2");
			
			break;
		}case 3:{
			// configuraci�n de la ejecuci�n
			int chunk2 = 32;
			dim3 tamGrid(1, N); //Grid dimensi�n
			dim3 tamBlock(chunk2,1,1); //Block dimensi�n
			// lanzamiento del kernel
			MatrixSumKernel_3<<<tamGrid, tamBlock>>>(M, A_dev, SumPar_dev);
			hipDeviceSynchronize();
			clockStop("kernel 3");
			
			break;
		}case 4:{
			// configuraci�n de la ejecuci�n
			int chunk2 = 16;
			dim3 tamGrid(N/chunk2, N/chunk2); //Grid dimensi�n
			dim3 tamBlock(chunk2,1,1); //Block dimensi�n
			// lanzamiento del kernel
			MatrixSumKernel_4<<<tamGrid, tamBlock>>>(M, A_dev, SumPar_dev);
			hipDeviceSynchronize();
			clockStop("kernel 4");
	
			break;
		}case 5:{
			// configuraci�n de la ejecuci�n
			int chunk2 = 16;
			dim3 tamGrid(N/chunk2, N/chunk2); //Grid dimensi�n
			dim3 tamBlock(chunk2,1,1); //Block dimensi�n
			// lanzamiento del kernel
			MatrixSumKernel_5<<<tamGrid, tamBlock>>>(M, A_dev, SumPar_dev);
			hipDeviceSynchronize();
			clockStop("kernel 5");
	
			break;
		}case 6:{
			// configuraci�n de la ejecuci�n
			int chunk2 = 32;
			dim3 tamGrid(1, N); //Grid dimensi�n
			dim3 tamBlock(chunk2,1,1); //Block dimensi�n
			// lanzamiento del kernel
			MatrixSumKernel_6<<<tamGrid, tamBlock,chunk2>>>(M, A_dev, SumPar_dev);
			hipDeviceSynchronize();
			clockStop("kernel 6");
	
			break;
		}
	}

	// Traer resultado;
	//clockStart();
	hipMemcpy(SumPar_hst, SumPar_dev, size2, hipMemcpyDeviceToHost);
	//clockStop("transf CPU <- GPU");

	// Sumar el vector de resultados parciales;
	float total = 0.0;
	for (int i = 0; i<N ; i++) total += SumPar_hst[i];

	free(SumPar_hst);
	// Free matrices en device
	hipFree(A_dev); hipFree(SumPar_dev); 

	return total;
}

float ejecutarCPU(float * A, int N){
	clockStart();
	float result_ref = sum_matrix(A,N);
	clockStop("CPU");	
	return result_ref;
}


int main(int argc, char** argv){
	
	int n;
	int algo;
	
	float *A;
	
	if (argc < 3){
		if (argc < 2){
			printf("Sin Parametros, asume tama�o 1024\n");
			n=1024;
		}else{
			n= atoi(argv[1]);
		}
		
		A = (float *)malloc(n*n*sizeof(float));
		
		ejecutarCPU(A,n);
		sumaColMatriz(n,n,A,0);
		sumaColMatriz(n,n,A,1);
		sumaColMatriz(n,n,A,2);
		sumaColMatriz(n,n,A,3);
		sumaColMatriz(n,n,A,4);
		sumaColMatriz(n,n,A,5);
		sumaColMatriz(n,n,A,6);
		exit(0);
	}
	
	n= atoi(argv[1]);
	algo = atoi(argv[2]);
	
	A = (float *)malloc(n*n*sizeof(float));
	init_matrix(A,n);
	float result_ref = ejecutarCPU(A,n);
	float result_gpu = sumaColMatriz(n,n,A,algo);
	
	if (result_gpu == result_ref){
		printf("\n\nResultado OK!! :)\n\n\n");
	}else{
		printf("\nSegui participando\n\n");
	}
	printf("GPU -> %f \n",result_gpu);
	printf("CPU -> %f \n\n",result_ref);
	free(A);	

	return 0;
}


